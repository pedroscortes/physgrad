#include "hip/hip_runtime.h"
#include "simulation.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <cmath>
#include <iostream>
#include <chrono>

namespace physgrad {

// Error checking utility
void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        std::cerr << "CUDA Error at " << file << ":" << line
                  << " - " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

__global__ void computeForces(
    float* acc_x, float* acc_y, float* acc_z,
    const float* pos_x, const float* pos_y, const float* pos_z,
    const float* mass, int n, float epsilon_sq, float G)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float xi = pos_x[i];
    float yi = pos_y[i];
    float zi = pos_z[i];

    float ax = 0.0f;
    float ay = 0.0f;
    float az = 0.0f;

    #pragma unroll 4
    for (int j = 0; j < n; j++) {
        if (i == j) continue;

        float dx = pos_x[j] - xi;
        float dy = pos_y[j] - yi;
        float dz = pos_z[j] - zi;

        float dist_sq = dx*dx + dy*dy + dz*dz + epsilon_sq;
        float inv_dist = rsqrtf(dist_sq);
        float inv_dist_cube = inv_dist * inv_dist * inv_dist;

        float force_mag = G * mass[j] * inv_dist_cube;

        ax += force_mag * dx;
        ay += force_mag * dy;
        az += force_mag * dz;
    }

    acc_x[i] = ax;
    acc_y[i] = ay;
    acc_z[i] = az;
}

__global__ void integrateLeapfrog(
    float* pos_x, float* pos_y, float* pos_z,
    float* vel_x, float* vel_y, float* vel_z,
    const float* acc_x, const float* acc_y, const float* acc_z,
    int n, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    vel_x[i] += acc_x[i] * dt;
    vel_y[i] += acc_y[i] * dt;
    vel_z[i] += acc_z[i] * dt;

    pos_x[i] += vel_x[i] * dt;
    pos_y[i] += vel_y[i] * dt;
    pos_z[i] += vel_z[i] * dt;
}

__global__ void initializeRandom(
    float* pos_x, float* pos_y, float* pos_z,
    float* vel_x, float* vel_y, float* vel_z,
    float* mass, int n,
    float cluster_scale, float velocity_scale,
    unsigned long long seed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    hiprandState_t state;
    hiprand_init(seed, i, 0, &state);

    pos_x[i] = hiprand_normal(&state) * cluster_scale;
    pos_y[i] = hiprand_normal(&state) * cluster_scale;
    pos_z[i] = hiprand_normal(&state) * cluster_scale;

    vel_x[i] = hiprand_normal(&state) * velocity_scale;
    vel_y[i] = hiprand_normal(&state) * velocity_scale;
    vel_z[i] = hiprand_normal(&state) * velocity_scale;

    mass[i] = 1.0f / n;
}

__global__ void packPositions(
    float* packed,
    const float* pos_x, const float* pos_y, const float* pos_z,
    const float* mass, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    packed[i * 4 + 0] = pos_x[i];
    packed[i * 4 + 1] = pos_y[i];
    packed[i * 4 + 2] = pos_z[i];
    packed[i * 4 + 3] = mass[i];
}

BodySystem::BodySystem(int num_bodies) : n(num_bodies) {
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        std::cerr << "Failed to set CUDA device: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    size_t size = n * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_pos_x, size));
    CUDA_CHECK(hipMalloc(&d_pos_y, size));
    CUDA_CHECK(hipMalloc(&d_pos_z, size));

    CUDA_CHECK(hipMalloc(&d_vel_x, size));
    CUDA_CHECK(hipMalloc(&d_vel_y, size));
    CUDA_CHECK(hipMalloc(&d_vel_z, size));

    CUDA_CHECK(hipMalloc(&d_acc_x, size));
    CUDA_CHECK(hipMalloc(&d_acc_y, size));
    CUDA_CHECK(hipMalloc(&d_acc_z, size));

    CUDA_CHECK(hipMalloc(&d_mass, size));

    CUDA_CHECK(hipMemset(d_acc_x, 0, size));
    CUDA_CHECK(hipMemset(d_acc_y, 0, size));
    CUDA_CHECK(hipMemset(d_acc_z, 0, size));
}

BodySystem::~BodySystem() {
    hipFree(d_pos_x);
    hipFree(d_pos_y);
    hipFree(d_pos_z);
    hipFree(d_vel_x);
    hipFree(d_vel_y);
    hipFree(d_vel_z);
    hipFree(d_acc_x);
    hipFree(d_acc_y);
    hipFree(d_acc_z);
    hipFree(d_mass);

    hipFree(d_grad_pos_x);
    hipFree(d_grad_pos_y);
    hipFree(d_grad_pos_z);
    hipFree(d_grad_vel_x);
    hipFree(d_grad_vel_y);
    hipFree(d_grad_vel_z);
}

void BodySystem::initializeCluster(const SimParams& params) {
    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    unsigned long long seed = std::chrono::system_clock::now().time_since_epoch().count();

    initializeRandom<<<blocks, threads>>>(
        d_pos_x, d_pos_y, d_pos_z,
        d_vel_x, d_vel_y, d_vel_z,
        d_mass, n,
        params.cluster_scale, params.velocity_scale,
        seed
    );

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

void BodySystem::getPositions(std::vector<float>& pos_x,
                              std::vector<float>& pos_y,
                              std::vector<float>& pos_z) const {
    pos_x.resize(n);
    pos_y.resize(n);
    pos_z.resize(n);

    size_t size = n * sizeof(float);
    CUDA_CHECK(hipMemcpy(pos_x.data(), d_pos_x, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(pos_y.data(), d_pos_y, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(pos_z.data(), d_pos_z, size, hipMemcpyDeviceToHost));
}

float BodySystem::computeEnergy(const SimParams& params) const {
    // For now, just compute kinetic energy as a simple validation
    // We'll add potential energy calculation later

    std::vector<float> vel_x(n), vel_y(n), vel_z(n), mass(n);

    size_t size = n * sizeof(float);
    CUDA_CHECK(hipMemcpy(vel_x.data(), d_vel_x, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(vel_y.data(), d_vel_y, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(vel_z.data(), d_vel_z, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(mass.data(), d_mass, size, hipMemcpyDeviceToHost));

    float kinetic_energy = 0.0f;
    for (int i = 0; i < n; i++) {
        float v2 = vel_x[i]*vel_x[i] + vel_y[i]*vel_y[i] + vel_z[i]*vel_z[i];
        kinetic_energy += 0.5f * mass[i] * v2;
    }

    return kinetic_energy;
}

void BodySystem::allocateGradients() {
    size_t size = n * sizeof(float);

    if (!d_grad_pos_x) {
        CUDA_CHECK(hipMalloc(&d_grad_pos_x, size));
        CUDA_CHECK(hipMalloc(&d_grad_pos_y, size));
        CUDA_CHECK(hipMalloc(&d_grad_pos_z, size));
        CUDA_CHECK(hipMalloc(&d_grad_vel_x, size));
        CUDA_CHECK(hipMalloc(&d_grad_vel_y, size));
        CUDA_CHECK(hipMalloc(&d_grad_vel_z, size));
    }
}

void BodySystem::zeroGradients() {
    if (!d_grad_pos_x) return;

    size_t size = n * sizeof(float);
    CUDA_CHECK(hipMemset(d_grad_pos_x, 0, size));
    CUDA_CHECK(hipMemset(d_grad_pos_y, 0, size));
    CUDA_CHECK(hipMemset(d_grad_pos_z, 0, size));
    CUDA_CHECK(hipMemset(d_grad_vel_x, 0, size));
    CUDA_CHECK(hipMemset(d_grad_vel_y, 0, size));
    CUDA_CHECK(hipMemset(d_grad_vel_z, 0, size));
}

void BodySystem::setGradientFromEnergy(float grad_energy) {
    allocateGradients();
    zeroGradients();

    float grad_value = grad_energy / n;
    std::vector<float> temp_grad(n, grad_value);

    size_t size = n * sizeof(float);
    CUDA_CHECK(hipMemcpy(d_grad_pos_x, temp_grad.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_grad_pos_y, temp_grad.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_grad_pos_z, temp_grad.data(), size, hipMemcpyHostToDevice));
}

void BodySystem::getGradients(std::vector<float>& grad_pos_x,
                             std::vector<float>& grad_pos_y,
                             std::vector<float>& grad_pos_z) const {
    if (!d_grad_pos_x) return;

    grad_pos_x.resize(n);
    grad_pos_y.resize(n);
    grad_pos_z.resize(n);

    size_t size = n * sizeof(float);
    CUDA_CHECK(hipMemcpy(grad_pos_x.data(), d_grad_pos_x, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(grad_pos_y.data(), d_grad_pos_y, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(grad_pos_z.data(), d_grad_pos_z, size, hipMemcpyDeviceToHost));
}


Simulation::Simulation(const SimParams& params)
    : params(params), bodies(std::make_unique<BodySystem>(params.num_bodies))
{
    CUDA_CHECK(hipStreamCreate(&compute_stream));
    CUDA_CHECK(hipStreamCreate(&transfer_stream));
    CUDA_CHECK(hipMalloc(&d_packed_positions, params.num_bodies * 4 * sizeof(float)));
    bodies->initializeCluster(params);

}

void Simulation::step() {
    auto start = std::chrono::high_resolution_clock::now();

    launchComputeForces(
        bodies->d_acc_x, bodies->d_acc_y, bodies->d_acc_z,
        bodies->d_pos_x, bodies->d_pos_y, bodies->d_pos_z,
        bodies->d_mass, bodies->n,
        params.epsilon * params.epsilon,
        params.G,
        compute_stream
    );

    launchIntegrate(
        bodies->d_pos_x, bodies->d_pos_y, bodies->d_pos_z,
        bodies->d_vel_x, bodies->d_vel_y, bodies->d_vel_z,
        bodies->d_acc_x, bodies->d_acc_y, bodies->d_acc_z,
        bodies->n, params.time_step,
        compute_stream
    );

    CUDA_CHECK(hipStreamSynchronize(compute_stream));

    auto end = std::chrono::high_resolution_clock::now();
    last_step_ms = std::chrono::duration<float, std::milli>(end - start).count();
}

float* Simulation::getPackedPositions() {
    int threads = 256;
    int blocks = (bodies->n + threads - 1) / threads;

    packPositions<<<blocks, threads, 0, transfer_stream>>>(
        d_packed_positions,
        bodies->d_pos_x, bodies->d_pos_y, bodies->d_pos_z,
        bodies->d_mass, bodies->n
    );

    CUDA_CHECK(hipStreamSynchronize(transfer_stream));
    return d_packed_positions;
}

float Simulation::getGFLOPS() const {
    float flops = 20.0f * bodies->n * (bodies->n - 1);
    return (flops / last_step_ms) / 1e6f;
}


void launchComputeForces(float* d_acc_x, float* d_acc_y, float* d_acc_z,
                        const float* d_pos_x, const float* d_pos_y, const float* d_pos_z,
                        const float* d_mass, int n, float epsilon, float G,
                        hipStream_t stream)
{
    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    computeForces<<<blocks, threads, 0, stream>>>(
        d_acc_x, d_acc_y, d_acc_z,
        d_pos_x, d_pos_y, d_pos_z,
        d_mass, n, epsilon, G
    );

    CUDA_CHECK(hipGetLastError());
}

void launchIntegrate(float* d_pos_x, float* d_pos_y, float* d_pos_z,
                    float* d_vel_x, float* d_vel_y, float* d_vel_z,
                    const float* d_acc_x, const float* d_acc_y, const float* d_acc_z,
                    int n, float dt, hipStream_t stream)
{
    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    integrateLeapfrog<<<blocks, threads, 0, stream>>>(
        d_pos_x, d_pos_y, d_pos_z,
        d_vel_x, d_vel_y, d_vel_z,
        d_acc_x, d_acc_y, d_acc_z,
        n, dt
    );

    CUDA_CHECK(hipGetLastError());
}

} // namespace physgrad