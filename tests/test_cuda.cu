#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "Found " << deviceCount << " CUDA devices" << std::endl;

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, i);
        if (err == hipSuccess) {
            std::cout << "Device " << i << ": " << prop.name << std::endl;
            std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
            std::cout << "  Total global memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
        }
    }

    // Test simple kernel
    float *d_test;
    err = hipMalloc(&d_test, 1024 * sizeof(float));
    if (err == hipSuccess) {
        std::cout << "CUDA malloc test: SUCCESS" << std::endl;
        hipFree(d_test);
    } else {
        std::cout << "CUDA malloc test: FAILED - " << hipGetErrorString(err) << std::endl;
    }

    return 0;
}